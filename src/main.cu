
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloFromGPU() {
    printf("Hello from GPU!\n");
}

int main() {
    helloFromGPU<<<1, 1>>>();
    hipDeviceSynchronize();
    std::cout << "Hello from CPU!\n";
    return 0;
}
